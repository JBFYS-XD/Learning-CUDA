#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "../tester/utils.h"

/**
 * @brief Find the k-th largest element in a vector using CUDA.
 * 
 * @tparam T Type of elements in the input vector (should support `int` and `float`).
 * @param h_input Host-side input vector.
 * @param k 1-based index of the element to find (e.g., `k=1` returns the largest element).
 * @return T The k-th largest element in `h_input`.

 * @note Must use CUDA kernels for all compute-intensive steps; no significant CPU allowed.
 * @note Library functions that can directly complete a significant part of the work are NOT allowed. 
 * @note For invalid cases, return T(-100).
 * @note Handles device memory management (allocate/copy/free) internally. Errors should be thrown.
 */

template <typename T>
__global__ void kTopSort1(T* input, int n) {
  extern __shared__ uint8_t shared_mem[];  // 单一符号，无类型冲突
  
  int blockSize = blockDim.x;
  // 手动划分：前 512 个 T 为 smem，后 512 个为 tmp
  T* smem = reinterpret_cast<T*>(shared_mem);
  T* tmp  = smem + (blockSize << 1);  // 指向后半部分
  
  int tid = threadIdx.x;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int posb = tid << 1, posg = idx << 1;
  
  smem[posb] = posg < n ? input[posg] : 1e9;
  smem[posb | 1] = (posg | 1) < n ? input[posg | 1] : 1e9;
  __syncthreads();
  for (int flag = 1; flag <= blockSize; flag <<= 1) {
    int start = posb;
    int mid = min(start + flag, blockSize << 1);
    int end = min(start + (flag << 1), blockSize << 1);
    if ((tid % flag) == 0) {
      int l = start, r = mid, k = start;
      while (l < mid && r < end) {
        if (smem[l] < smem[r])
        tmp[k ++] = smem[l ++];
        else
        tmp[k ++] = smem[r ++];
      }
      
      while (l < mid) tmp[k ++] = smem[l ++];
      while (r < end) tmp[k ++] = smem[r ++];
      
    }
    __syncthreads();
    
    smem[start] = tmp[start];
    smem[start + 1] = tmp[start + 1];
    __syncthreads();
  }
  
  if (posg < n) input[posg] = smem[posb];
  if ((posg | 1) < n) input[posg | 1] = smem[posb | 1];
}

template <typename T>
__global__ void kTopSort2(T* input, int n, T* gtmp, int flag) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int start = bid * flag * 2;
  int mid = min(start + flag, n);
  int end = min(start + (flag << 1), n);

  if (tid == 0) {
    int l = start, r = mid, k = start;
    while (l < mid && r < end) {
      if (input[l] < input[r])
        gtmp[k ++] = input[l ++];
      else
        gtmp[k ++] = input[r ++];
    }
  
    while (l < mid) gtmp[k ++] = input[l ++];
    while (r < end) gtmp[k ++] = input[r ++];
  }

  __syncthreads();
  for (int i = start + tid; i < end; i += blockDim.x) {
    if (i < n)
      input[i] = gtmp[i];
  }

}

template <typename T>
void kTopSort_Work(T* input, T* gtmp, size_t n) {
  int blockSize = 256;

  int flag = 1;
  int gridSize = ((n + flag - 1) / flag + blockSize - 1) / blockSize;
  size_t shared_mem = blockSize * sizeof(T) * 4;
  kTopSort1<T><<<gridSize, blockSize, shared_mem>>>(input, n);
  CUDA_CHECK(hipDeviceSynchronize());

  for (flag = 512; flag < n; flag <<= 1) {
    gridSize = (n + flag - 1) / flag;
    kTopSort2<T><<<gridSize, blockSize>>>(input, n, gtmp, flag);
  }
  CUDA_CHECK(hipDeviceSynchronize());
}

template <typename T>
T kthLargest(const std::vector<T>& h_input, size_t k) {
  // TODO: Implement the kthLargest function
  
  size_t n = h_input.size();
  if (k < 1 || k > n) return T(-100);
  
  size_t size_T = sizeof(T);
  size_t size_arr = n * size_T;
  
  T *d_input;
  T* gtmp;
  CUDA_CHECK(hipMalloc(&d_input, size_arr));
  CUDA_CHECK(hipMalloc(&gtmp, n * sizeof(T)));
  CUDA_CHECK(hipMemcpy(d_input, h_input.data(), size_arr, hipMemcpyHostToDevice));
  
  kTopSort_Work<T>(d_input, gtmp, n);
  
  T result;
  CUDA_CHECK(hipMemcpy(&result, d_input + (n - k), size_T, hipMemcpyDeviceToHost));
  
  CUDA_CHECK(hipFree(gtmp));
  CUDA_CHECK(hipFree(d_input));
  return result;
}

/**
 * @brief Computes flash attention for given query, key, and value tensors.
 * 
 * @tparam T Data type (float) for input/output tensors
 * @param[in] h_q Query tensor of shape [batch_size, tgt_seq_len, query_heads, head_dim]
 * @param[in] h_k Key tensor of shape [batch_size, src_seq_len, kv_heads, head_dim]
 * @param[in] h_v Value tensor of shape [batch_size, src_seq_len, kv_heads, head_dim]
 * @param[out] h_o Output attention tensor of shape [batch_size, tgt_seq_len, query_heads, head_dim]
 * @param[in] batch_size Batch dimension size
 * @param[in] target_seq_len Target sequence length
 * @param[in] src_seq_len Source sequence length  
 * @param[in] query_heads Number of query attention heads
 * @param[in] kv_heads Number of key/value heads (supports grouped query attention)
 * @param[in] head_dim Dimension size of each attention head
 * @param[in] is_causal Whether to apply causal masking
 */

template <typename T>
__device__ T block_reduce_max(T* smem, T val) {
  int tid = threadIdx.x;
  int lane = tid % 32;
  int warp_id = tid / 32;

  for (int offset = 16; offset > 0; offset >>= 1) {
    T other = __shfl_xor_sync(0xFFFFFFFF, val, offset);
    val = fmaxf(val, other);
  }

  if (lane == 0)
    smem[warp_id] = val;
  __syncthreads();

  if (warp_id == 0) {
    val = lane >= 8 ? -1e9 : smem[lane];
    for (int offset = 16; offset > 0; offset >>= 1) {
      T other = __shfl_xor_sync(0xFFFFFFFF, val, offset);
      val = fmaxf(val, other);
    }
    smem[0] = val;
  }
  __syncthreads();

  return smem[0];
}

template <typename T>
__device__ T block_reduce_sum(T* smem, T val) {
  int tid = threadIdx.x;
  int lane = tid % 32;
  int warp_id = tid / 32;

  for (int offset = 16; offset > 0; offset >>= 1) {
    T other = __shfl_xor_sync(0xFFFFFFFF, val, offset);
    val = val + other;
  }

  if (lane == 0)
    smem[warp_id] = val;
  __syncthreads();

  if (warp_id == 0) {
    val = lane >= 8 ? 0 : smem[lane];
    for (int offset = 16; offset > 0; offset >>= 1) {
      T other = __shfl_xor_sync(0xFFFFFFFF, val, offset);
      val = val + other;
    }
    smem[0] = val;
  }
  __syncthreads();

  return smem[0];
}

template <typename T>
__global__ void flashAttentionKernel(
  const T* query, const T* key,
  const T* value, T* output,
  int batch_size, int target_seq_len, int src_seq_len, 
  int query_heads, int kv_heads, int head_dim, bool is_causal
) {

  
  int batch_id = blockIdx.x;
  int heads_id = blockIdx.y;
  int kv_id = heads_id * kv_heads / query_heads;

  if (batch_id >= batch_size || heads_id >= query_heads) return;
  
  extern __shared__ uint8_t shared_mem[];
  T* scores = reinterpret_cast<T*>(shared_mem);
  T* smem = scores + src_seq_len;

  for (int tgt = 0; tgt < target_seq_len; tgt ++) {
    float mx = -1e9;
    for (int src = threadIdx.x; src < src_seq_len; src += blockDim.x) {
      if (is_causal && tgt < src) {
        scores[src] = -1e9f;
      } else {
        float sum = 0.;
        for (int dim = 0; dim < head_dim; dim ++) {
          // query[batch_id][tgt][heads_id][dim]
          int qid = dim + head_dim * (heads_id + query_heads * (tgt + target_seq_len * batch_id));
          // key[batch_id][src][kv_id][dim]
          int kid = dim + head_dim * (kv_id + kv_heads * (src + src_seq_len * batch_id));
          sum += query[qid] * key[kid];
        }
        scores[src] = sum / sqrtf(float(head_dim));
        mx = fmaxf(mx, scores[src]);
      }
    }
    __syncthreads();

    mx = block_reduce_max(smem, mx);
    
    T sum = 0.;
    for (int src = threadIdx.x; src < src_seq_len; src += blockDim.x) {
      scores[src] = expf(scores[src] - mx);
      sum += scores[src];
    }
    __syncthreads();

    sum = block_reduce_sum(smem, sum);

    for (int src = threadIdx.x; src < src_seq_len; src += blockDim.x) {
      scores[src] = scores[src] / (sum + 1e-8f);
    }
    __syncthreads();

    for (int dim = threadIdx.x; dim < head_dim; dim += blockDim.x) {
      float sum = 0.;
      for (int src = 0; src < src_seq_len; src ++) {
        // value[batch_id][src][kv_id][dim]
        int vidx = dim + head_dim * (kv_id + kv_heads * (src + src_seq_len * batch_id));
        sum += scores[src] * value[vidx];
      }
      // output[batch_id][tgt][heads_id][dim]
      int oidx = dim + head_dim * (heads_id + query_heads * (tgt + target_seq_len * batch_id));
      output[oidx] = sum;
    }
    __syncthreads();
  }
}


template <typename T>
void flashAttention(
  const std::vector<T>& h_q, const std::vector<T>& h_k,
  const std::vector<T>& h_v, std::vector<T>& h_o,
  int batch_size, int target_seq_len, int src_seq_len, 
  int query_heads, int kv_heads, int head_dim, bool is_causal
) {

  h_o.resize(batch_size * target_seq_len * query_heads * head_dim);

  T* d_q, *d_k, *d_v, *d_o;
  size_t size_q = h_q.size() * sizeof(T);
  size_t size_k = h_k.size() * sizeof(T);
  size_t size_v = h_v.size() * sizeof(T);
  size_t size_o = h_o.size() * sizeof(T);
  
  CUDA_CHECK(hipMalloc(&d_q, size_q));
  CUDA_CHECK(hipMalloc(&d_k, size_k));
  CUDA_CHECK(hipMalloc(&d_v, size_v));
  CUDA_CHECK(hipMalloc(&d_o, size_o));

  CUDA_CHECK(hipMemcpy(d_q, h_q.data(), size_q, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_k, h_k.data(), size_k, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_v, h_v.data(), size_v, hipMemcpyHostToDevice));

  dim3 gridSize(batch_size, query_heads);
  int blockSize = 256;
  size_t shared_mem = src_seq_len * sizeof(T) + 8 * sizeof(T);
  flashAttentionKernel<T><<<gridSize, blockSize, shared_mem>>>(
    d_q, d_k, d_v, d_o,
    batch_size, target_seq_len, src_seq_len, 
    query_heads, kv_heads, head_dim, is_causal
  );

  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(h_o.data(), d_o, size_o, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_q));
  CUDA_CHECK(hipFree(d_k));
  CUDA_CHECK(hipFree(d_v));
  CUDA_CHECK(hipFree(d_o));

}

// *********************************************************************
// Explicit Template Instantiations (REQUIRED FOR LINKING WITH TESTER.O)
// DO NOT MODIFY THIS SECTION
// *********************************************************************
template int kthLargest<int>(const std::vector<int>&, size_t);
template float kthLargest<float>(const std::vector<float>&, size_t);
template void flashAttention<float>(const std::vector<float>&, const std::vector<float>&,
  const std::vector<float>&, std::vector<float>&,
  int, int, int, int, int, int, bool);
